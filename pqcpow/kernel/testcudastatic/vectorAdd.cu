#include <hip/hip_runtime.h>
#include <iostream>
__global__ void vectorAddKernel(float* A, float* B, float* C, int N) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < N) {
        C[i] = A[i] + B[i];
    }
}
 
extern "C" {
    int getDeviceCount()
    {
        int deviceCount = 0;
        hipGetDeviceCount(&deviceCount);
        printf("Device count:%d", deviceCount);
        return deviceCount;
    }
    void vectorAdd(float* A, float* B, float* C, int N) {
        float* devA;
        float* devB;
        float* devC;
        hipMalloc((void**)&devA, N * sizeof(float));
        hipMalloc((void**)&devB, N * sizeof(float));
        hipMalloc((void**)&devC, N * sizeof(float));
        hipMemcpy(devA, A, N * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(devB, B, N * sizeof(float), hipMemcpyHostToDevice);
 
        int blockSize = 256;
        int numBlocks = (N + blockSize - 1) / blockSize;
        vectorAddKernel<<<numBlocks, blockSize>>>(devA, devB, devC, N);
 
        hipMemcpy(C, devC, N * sizeof(float), hipMemcpyDeviceToHost);
        hipFree(devA);
        hipFree(devB);
        hipFree(devC);
    }
}